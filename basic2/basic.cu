// GPU/CPU
__host__ __device__
static void vadd_calc(float a, float b, float *c)
{
	*c = a + b;
}

// GPU
__global__
static void vadd_gpu(int n, const float *a, const float *b, float *c)
{
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	if (tid < n) {
		vadd_calc(a[tid], b[tid], &c[tid]);
	}
}

// CPU
static void vadd_cpu(int n, const float *a, const float *b, float *c)
{
	for (int i = 0; i < n; i++) {
		vadd_calc(a[i], b[i], &c[i]);
	}
}

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

static void divide_array(const int, const int, int [], int []);

int main(int argc, char **argv)
{
	int    gpu = 1;
	int    um = 0;
	int    ngpu = 1;
	int    nvector = 1000;
	int    nloop = 1000;
	int    *offset, *length;
	size_t size;
#ifndef _OPENMP
	hipStream_t *stream;
#endif

	// arguments
	while (--argc) {
		argv++;
		if      (!strcmp(*argv, "-gpu")) {
			gpu = 1;
		}
		else if (!strcmp(*argv, "-cpu")) {
			gpu = 0;
		}
		else if (!strcmp(*argv, "-hdm")) {
			um = 0;
		}
		else if (!strcmp(*argv, "-um")) {
			um = 1;
		}
		else if (argc == 3) {
			ngpu = atoi(*argv);
			if (ngpu < 1) ngpu = 1;
		}
		else if (argc == 2) {
			nvector = atoi(*argv);
			if (nvector < 1) nvector = 1;
		}
		else if (argc == 1) {
			nloop = atoi(*argv);
			if (nloop < 1) nloop = 1;
		}
	}

	// setup GPU
	if (gpu) {
		// check device
		int ndevice;
		hipGetDeviceCount(&ndevice);
		if (ndevice < 1) {
			printf("No CUDA device.\n");
			exit(1);
		}

		// GPU info
		for (int igpu = 0; igpu < ngpu; igpu++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, igpu);
			printf("GPU-%d : %s, C.C.%d.%d, U.M.%s\n", igpu, prop.name, prop.major, prop.minor, (um ? "ON" : "OFF"));
		}
	}

	// host memory
	size = nvector * sizeof(float);
	float *h_a = (float *)malloc(size);
	float *h_b = (float *)malloc(size);
	float *h_c = (float *)malloc(size);

	// offset and length
	size = ngpu * sizeof(int);
	offset = (int *)malloc(size);
	length = (int *)malloc(size);
	divide_array(nvector, ngpu, offset, length);

	// device memory
	size = ngpu * sizeof(float *);
	float **d_a = (float **)malloc(size);
	float **d_b = (float **)malloc(size);
	float **d_c = (float **)malloc(size);
	for (int igpu = 0; igpu < ngpu; igpu++) {
		if (gpu) hipSetDevice(igpu);  // 重要
		size = length[igpu] * sizeof(float);
		hipMalloc((void **)&d_a[igpu], size);
		hipMalloc((void **)&d_b[igpu], size);
		hipMalloc((void **)&d_c[igpu], size);
	}

	// setup problem
	for (int i = 0; i < nvector; i++) {
		h_a[i] = i;
		h_b[i] = i + 1;
		h_c[i] = 0;
	}

	// copy host to device
	for (int igpu = 0; igpu < ngpu; igpu++) {
		size = length[igpu] * sizeof(float);
		hipMemcpy(d_a[igpu], h_a + offset[igpu], size, hipMemcpyHostToDevice);
		hipMemcpy(d_b[igpu], h_b + offset[igpu], size, hipMemcpyHostToDevice);
	}

	// timer
	for (int igpu = 0; igpu < ngpu; igpu++) {
		hipSetDevice(igpu);
		hipDeviceSynchronize();
	}
	clock_t t0 = clock();

	// calculation
	for (int loop = 0; loop < nloop; loop++) {
		if (gpu) {
			// GPU
			// streams
			for (int igpu = 0; igpu < ngpu; igpu++) {
				hipSetDevice(igpu);
				int block = 256;
				int grid = (length[igpu] + (block - 1)) / block;
				vadd_gpu<<<grid, block, 0, stream[igpu]>>>(length[igpu], d_a[igpu], d_b[igpu], d_c[igpu]);
			}

			// synchronize streams
			for (int igpu = 0; igpu < ngpu; igpu++) {
				hipSetDevice(igpu);
				hipStreamSynchronize(stream[igpu]);
			}
		}
		else {
			// CPU
			for (int igpu = 0; igpu < ngpu; igpu++) {
				vadd_cpu(length[igpu], h_a + offset[igpu], h_b + offset[igpu], h_c + offset[igpu]);
			}
		}
	}

	// timer
	for (int igpu = 0; igpu < ngpu; igpu++) {
		hipSetDevice(igpu);
		hipDeviceSynchronize();
	}
	clock_t t1 = clock();

	// copy device to host
	for (int igpu = 0; igpu < ngpu; igpu++) {
		hipMemcpy(h_c + offset[igpu], d_c[igpu], size, hipMemcpyDeviceToHost);
	}

	// sum
	double sum = 0;
	for (int i = 0; i < nvector; i++) {
		sum += h_c[i];
	}

	// output
	double exact = (double)nvector * nvector;
	double sec = (double)(t1 - t0) / CLOCKS_PER_SEC;
	printf("nvector=%d nloop=%d %e(%e) %s[sec]=%.3f\n",
		nvector, nloop, sum, exact, (gpu ? "GPU" : "CPU"), sec);

	// free
  /*
	for (int igpu = 0; igpu < ngpu; igpu++) {
		cudaFree(gpu, d_a[igpu]);
		cudaFree(gpu, d_b[igpu]);
		cudaFree(gpu, d_c[igpu]);
	}*/

	free(h_a);
	free(h_b);
	free(h_c);
	free(d_a);
	free(d_b);
	free(d_c);
	hipDeviceReset();

	return 0;
}

static void divide_array(const int n, const int ndiv, int offset[], int length[])
{
	int l_n = (n + (ndiv - 1)) / ndiv;
	for (int i = 0; i < ndiv; i++) {
		offset[i] = i * l_n;
		int istop = (i + 1) * l_n;
		if (istop > n) istop = n;
		length[i] = istop - offset[i];
	}
}
